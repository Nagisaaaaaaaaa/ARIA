#include "hip/hip_runtime.h"
#include "ARIA/BezierCurve.h"

#include <gtest/gtest.h>

namespace ARIA {

TEST(BezierCurve, Base) {
  static_assert(MovingPoint<BezierCurve, float, 1, NonRational, DegreeDynamic, std::vector<Vec1f>>);
  static_assert(MovingPoint<BezierCurve, float, 2, NonRational, DegreeDynamic, std::vector<Vec2f>>);
  static_assert(MovingPoint<BezierCurve, float, 3, NonRational, DegreeDynamic, std::vector<Vec3f>>);

  static_assert(MovingPoint<BezierCurve, float, 1, Rational, DegreeDynamic, std::vector<Vec2f>>);
  static_assert(MovingPoint<BezierCurve, float, 2, Rational, DegreeDynamic, std::vector<Vec3f>>);
  static_assert(MovingPoint<BezierCurve, float, 3, Rational, DegreeDynamic, std::vector<Vec4f>>);

  static_assert(MovingPoint<BezierCurve, float, 1, NonRational, Degree<2>, std::vector<Vec1f>>);
  static_assert(MovingPoint<BezierCurve, float, 2, NonRational, Degree<2>, std::vector<Vec2f>>);
  static_assert(MovingPoint<BezierCurve, float, 3, NonRational, Degree<2>, std::vector<Vec3f>>);

  static_assert(MovingPoint<BezierCurve, float, 1, Rational, Degree<2>, std::vector<Vec2f>>);
  static_assert(MovingPoint<BezierCurve, float, 2, Rational, Degree<2>, std::vector<Vec3f>>);
  static_assert(MovingPoint<BezierCurve, float, 3, Rational, Degree<2>, std::vector<Vec4f>>);

  static_assert(MovingPoint<BezierCurve, float, 1, NonRational, Degree<3>, std::vector<Vec1f>>);
  static_assert(MovingPoint<BezierCurve, float, 2, NonRational, Degree<3>, std::vector<Vec2f>>);
  static_assert(MovingPoint<BezierCurve, float, 3, NonRational, Degree<3>, std::vector<Vec3f>>);

  static_assert(MovingPoint<BezierCurve, float, 1, Rational, Degree<3>, std::vector<Vec2f>>);
  static_assert(MovingPoint<BezierCurve, float, 2, Rational, Degree<3>, std::vector<Vec3f>>);
  static_assert(MovingPoint<BezierCurve, float, 3, Rational, Degree<3>, std::vector<Vec4f>>);
}

TEST(BezierCurve, NonRational3D) {
  auto expectSphere = [](const auto &bezier) {
    for (float t = 0; t <= 1; t += 0.01) {
      Vec3f pHomo = bezier(t);
      Vec2f p = Vec2f(pHomo.x(), pHomo.y()) / pHomo.z();
      EXPECT_FLOAT_EQ(p.norm(), 1);
    }
  };

  // Static degree + std::vector.
  {
    std::vector<Vec3f> controlPoints = {{1, 0, 1}, {1, 1, 1}, {0, 2, 2}};
    BezierCurve<float, 3, NonRational, Degree<2>, std::vector<Vec3f>> bezier{controlPoints};
    expectSphere(bezier);
  }

  // Static degree + std::array.
  {
    std::array<Vec3f, 3> controlPoints = {Vec3f{1, 0, 1}, {1, 1, 1}, {0, 2, 2}};
    BezierCurve<float, 3, NonRational, Degree<2>, std::array<Vec3f, 3>> bezier{controlPoints};
    expectSphere(bezier);
  }

  // Dynamic degree + std::vector
  {
    std::vector<Vec3f> controlPoints = {{1, 0, 1}, {1, 1, 1}, {0, 2, 2}};
    BezierCurve<float, 3, NonRational, DegreeDynamic, std::vector<Vec3f>> bezier{controlPoints};
    expectSphere(bezier);
  }

  // Dynamic degree + std::array
  {
    std::array<Vec3f, 3> controlPoints = {Vec3f{1, 0, 1}, {1, 1, 1}, {0, 2, 2}};
    BezierCurve<float, 3, NonRational, DegreeDynamic, std::array<Vec3f, 3>> bezier{controlPoints};
    expectSphere(bezier);
  }
}

} // namespace ARIA
