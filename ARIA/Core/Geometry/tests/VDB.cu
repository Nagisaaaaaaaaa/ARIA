#include "hip/hip_runtime.h"
#include "ARIA/VDB.h"

#include <gtest/gtest.h>

namespace ARIA {

namespace {

struct PatternInts {
  int v[2];
};

struct PatternFloats {
  float v[2];
};

} // namespace

template <>
struct Mosaic<int, PatternInts> {
  ARIA_HOST_DEVICE PatternInts operator()(const int &v) const { return {.v = {v / 2, v % 2}}; }

  ARIA_HOST_DEVICE int operator()(const PatternInts &v) const { return v.v[0] * 2 + v.v[1]; }
};

template <>
struct Mosaic<float, PatternFloats> {
  ARIA_HOST_DEVICE PatternFloats operator()(const float &v) const {
    return {.v = {v * (2.0F / 5.0F), v * (3.0F / 5.0F)}};
  }

  ARIA_HOST_DEVICE float operator()(const PatternFloats &v) const { return v.v[0] + v.v[1]; }
};

namespace {

template <typename T_, typename U_>
ARIA_HOST_DEVICE inline void AssertEq(const T_ &a_, const U_ &b_) {
  auto a = Auto(a_);
  auto b = Auto(b_);
  using T = decltype(a);
  using U = decltype(b);

  if constexpr (std::integral<T> && std::integral<U>) {
    ARIA_ASSERT(a == b);
  } else {
    double aD = a;
    double bD = b;

    double threshold0 = 1e-6;
    double threshold1 = std::max(std::abs(aD), std::abs(bD)) / 1000.0;
    double threshold = std::max(threshold0, threshold1);

    ARIA_ASSERT(std::abs(aD - bD) < threshold);
  }
}

template <typename T>
void Test1DVDBHandleKernels() {
  using Handle = vdb::detail::VDBHandle<T, 1, SpaceDevice>;

  const int n = 20000;
  const int nHalf = n / 2;

  // Dense accesses.
  {
    Handle handle = Handle::Create();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      handle.value_AllocateIfNotExist(Vec1i{i - nHalf}) = i - nHalf;
    }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(handle.value_AllocateIfNotExist(Vec1i{i - nHalf}), i - nHalf);
    }).Launch();
    cuda::device::current::get().synchronize();
    handle.Destroy();
  }

  // Checkerboard accesses.
  {
    Handle handle = Handle::Create();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      handle.value_AllocateIfNotExist(Vec1i{i - nHalf} * 2) = nHalf - i;
    }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(handle.value_AssumeExist(Vec1i{i - nHalf} * 2), nHalf - i);
    }).Launch();
    cuda::device::current::get().synchronize();
    handle.Destroy();
  }
}

template <typename T>
void Test2DVDBHandleKernels() {
  using Handle = vdb::detail::VDBHandle<T, 2, SpaceDevice>;

  const Layout layout = make_layout_major(200, 300);
  const int n = 20000;
  const int nHalf = n / 2;

  // Dense accesses.
  {
    Handle handle = Handle::Create();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      handle.value_AllocateIfNotExist(ToVec(tec)) = layout(tec);
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      AssertEq(handle.value_AllocateIfNotExist(ToVec(tec)), layout(tec));
    }).Launch();
    cuda::device::current::get().synchronize();
    handle.Destroy();
  }

  // Checkerboard accesses.
  {
    Handle handle = Handle::Create();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      if ((get<0>(tec) + get<1>(tec)) % 2 == 0)
        handle.value_AllocateIfNotExist(ToVec(tec)) = layout(tec);
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      if ((get<0>(tec) + get<1>(tec)) % 2 == 0)
        AssertEq(handle.value_AssumeExist(ToVec(tec)), layout(tec));
    }).Launch();
    cuda::device::current::get().synchronize();
    handle.Destroy();
  }

  // Sparse accesses, 1D.
  { // x.
    Handle handle = Handle::Create();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      handle.value_AllocateIfNotExist(Vec2i{i - nHalf, 0}) = i - nHalf;
    }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(handle.value_AllocateIfNotExist(Vec2i{i - nHalf, 0}), i - nHalf);
    }).Launch();
    cuda::device::current::get().synchronize();
    handle.Destroy();
  }
  { // y.
    Handle handle = Handle::Create();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      handle.value_AllocateIfNotExist(Vec2i{0, nHalf - i}) = i - nHalf;
    }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(handle.value_AssumeExist(Vec2i{0, nHalf - i}), i - nHalf);
    }).Launch();
    cuda::device::current::get().synchronize();
    handle.Destroy();
  }

  // Sparse accesses, 2D.
  {
    Handle handle = Handle::Create();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      handle.value_AllocateIfNotExist(Vec2i{i - nHalf, i - nHalf}) = i - nHalf;
      handle.value_AllocateIfNotExist(Vec2i{i - nHalf, nHalf - i}) = i - nHalf;
    }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(handle.value_AllocateIfNotExist(Vec2i{i - nHalf, i - nHalf}), i - nHalf);
      AssertEq(handle.value_AssumeExist(Vec2i{i - nHalf, nHalf - i}), i - nHalf);
    }).Launch();
    cuda::device::current::get().synchronize();
    handle.Destroy();
  }
}

template <typename T>
void Test3DVDBHandleKernels() {
  using Handle = vdb::detail::VDBHandle<T, 3, SpaceDevice>;

  const Layout layout = make_layout_major(50, 100, 150);
  const int n = 1000;
  const int nHalf = n / 2;

  // Dense accesses.
  {
    Handle handle = Handle::Create();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      handle.value_AllocateIfNotExist(ToVec(tec)) = layout(tec);
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      AssertEq(handle.value_AllocateIfNotExist(ToVec(tec)), layout(tec));
    }).Launch();
    cuda::device::current::get().synchronize();
    handle.Destroy();
  }

  // Checkerboard accesses.
  {
    Handle handle = Handle::Create();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      if ((get<0>(tec) + get<1>(tec) + get<2>(tec)) % 2 == 0)
        handle.value_AllocateIfNotExist(ToVec(tec)) = layout(tec);
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      if ((get<0>(tec) + get<1>(tec) + get<2>(tec)) % 2 == 0)
        AssertEq(handle.value_AssumeExist(ToVec(tec)), layout(tec));
    }).Launch();
    cuda::device::current::get().synchronize();
    handle.Destroy();
  }

  // Sparse accesses, 1D.
  { // x.
    Handle handle = Handle::Create();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      handle.value_AllocateIfNotExist(Vec3i{i - nHalf, 0, 0}) = i - nHalf;
    }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(handle.value_AllocateIfNotExist(Vec3i{i - nHalf, 0, 0}), i - nHalf);
    }).Launch();
    cuda::device::current::get().synchronize();
    handle.Destroy();
  }
  { // y.
    Handle handle = Handle::Create();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      handle.value_AllocateIfNotExist(Vec3i{0, nHalf - i, 0}) = i - nHalf;
    }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(handle.value_AssumeExist(Vec3i{0, nHalf - i, 0}), i - nHalf);
    }).Launch();
    cuda::device::current::get().synchronize();
    handle.Destroy();
  }
  { // z.
    Handle handle = Handle::Create();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      handle.value_AllocateIfNotExist(Vec3i{0, 0, nHalf - i}) = i - nHalf;
    }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(handle.value_AssumeExist(Vec3i{0, 0, nHalf - i}), i - nHalf);
    }).Launch();
    cuda::device::current::get().synchronize();
    handle.Destroy();
  }

  // Sparse accesses, 3D.
  {
    Handle handle = Handle::Create();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      handle.value_AllocateIfNotExist(Vec3i{i - nHalf, i - nHalf, 0}) = i - nHalf;
      handle.value_AllocateIfNotExist(Vec3i{i - nHalf, nHalf - i, 0}) = i - nHalf;

      if (i - nHalf != nHalf - i) {
        handle.value_AllocateIfNotExist(Vec3i{i - nHalf, 0, i - nHalf}) = (i - nHalf) * 2;
        handle.value_AllocateIfNotExist(Vec3i{i - nHalf, 0, nHalf - i}) = (i - nHalf) * 2;
      }

      if (i - nHalf != nHalf - i) {
        handle.value_AllocateIfNotExist(Vec3i{0, i - nHalf, i - nHalf}) = (i - nHalf) * (-3);
        handle.value_AllocateIfNotExist(Vec3i{0, i - nHalf, nHalf - i}) = (i - nHalf) * (-3);
      }
    }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(handle.value_AllocateIfNotExist(Vec3i{i - nHalf, i - nHalf, 0}), i - nHalf);
      AssertEq(handle.value_AssumeExist(Vec3i{i - nHalf, nHalf - i, 0}), i - nHalf);

      if (i - nHalf != nHalf - i) {
        AssertEq(handle.value_AllocateIfNotExist(Vec3i{i - nHalf, 0, i - nHalf}), (i - nHalf) * 2);
        AssertEq(handle.value_AssumeExist(Vec3i{i - nHalf, 0, nHalf - i}), (i - nHalf) * 2);
      }

      if (i - nHalf != nHalf - i) {
        AssertEq(handle.value_AllocateIfNotExist(Vec3i{0, i - nHalf, i - nHalf}), (i - nHalf) * (-3));
        AssertEq(handle.value_AssumeExist(Vec3i{0, i - nHalf, nHalf - i}), (i - nHalf) * (-3));
      }
    }).Launch();
    cuda::device::current::get().synchronize();
    handle.Destroy();
  }
}

template <typename T>
void Test1DVDBKernels() {
  using V = DeviceVDB<T, 1>;
  using AllocateWriteAccessor = VDBAllocateWriteAccessor<V>;
  using WriteAccessor = VDBWriteAccessor<V>;
  using ReadAccessor = VDBReadAccessor<V>;

  const int n = 20000;
  const int nHalf = n / 2;

  // CTAD.
  {
    V v;
    VDBAccessor allocateWriteAccessor = v.allocateWriteAccessor();
    VDBAccessor writeAccessor = v.writeAccessor();
    VDBAccessor readAccessor = v.readAccessor();

    static_assert(std::is_same_v<decltype(allocateWriteAccessor), AllocateWriteAccessor>);
    static_assert(std::is_same_v<decltype(writeAccessor), WriteAccessor>);
    static_assert(std::is_same_v<decltype(readAccessor), ReadAccessor>);
  }

  // Dense accesses.
  {
    V v;
    VDBAccessor allocateWriteAccessor = v.allocateWriteAccessor();
    VDBAccessor writeAccessor = v.writeAccessor();
    VDBAccessor readAccessor = v.readAccessor();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable { allocateWriteAccessor.value(Tec{i - nHalf}) = i - nHalf; }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(allocateWriteAccessor.value(Tec{i - nHalf}), i - nHalf);
    }).Launch();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable { writeAccessor.value(Tec{i - nHalf}) *= -2; }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(writeAccessor.value(Tec{i - nHalf}), (i - nHalf) * (-2));
    }).Launch();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(readAccessor.value(Tec{i - nHalf}), (i - nHalf) * (-2));
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int> &tec) mutable { writeAccessor.value(tec) += 233; }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int> &tec) mutable {
      AssertEq(readAccessor.value(tec), get<0>(tec) * (-2) + 233);
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int> &tec, AllocateWriteAccessor accessor) mutable {
      accessor.value(tec) += 233;
    }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int> &tec, ReadAccessor accessor) mutable {
      AssertEq(accessor.value(tec), get<0>(tec) * (-2) + 466);
    }).Launch();
  }

  // Checkerboard accesses.
  {
    V v;
    VDBAccessor allocateWriteAccessor = v.allocateWriteAccessor();
    VDBAccessor writeAccessor = v.writeAccessor();
    VDBAccessor readAccessor = v.readAccessor();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      allocateWriteAccessor.value(Tec{i - nHalf} * 2) = nHalf - i;
    }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(allocateWriteAccessor.value(Tec{i - nHalf} * 2), nHalf - i);
    }).Launch();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable { writeAccessor.value(Tec{i - nHalf} * 2) *= -2; }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(writeAccessor.value(Tec{i - nHalf} * 2), (nHalf - i) * (-2));
    }).Launch();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(readAccessor.value(Tec{i - nHalf} * 2), (nHalf - i) * (-2));
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int> &tec) mutable { writeAccessor.value(tec) -= 233; }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int> &tec) mutable {
      AssertEq(readAccessor.value(tec), get<0>(tec) - 233);
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int> &tec, WriteAccessor &accessor) mutable {
      accessor.value(tec) -= 233;
    }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int> &tec, const ReadAccessor &accessor) mutable {
      AssertEq(accessor.value(tec), get<0>(tec) - 466);
    }).Launch();
  }

  cuda::device::current::get().synchronize();
}

template <typename T>
void Test2DVDBKernels() {
  using V = DeviceVDB<T, 2>;
  using AllocateWriteAccessor = VDBAllocateWriteAccessor<V>;
  using WriteAccessor = VDBWriteAccessor<V>;
  using ReadAccessor = VDBReadAccessor<V>;

  const Layout layout = make_layout_major(200, 300);
  const int n = 20000;
  const int nHalf = n / 2;

  // CTAD.
  {
    V v;
    VDBAccessor allocateWriteAccessor = v.allocateWriteAccessor();
    VDBAccessor writeAccessor = v.writeAccessor();
    VDBAccessor readAccessor = v.readAccessor();

    static_assert(std::is_same_v<decltype(allocateWriteAccessor), AllocateWriteAccessor>);
    static_assert(std::is_same_v<decltype(writeAccessor), WriteAccessor>);
    static_assert(std::is_same_v<decltype(readAccessor), ReadAccessor>);
  }

  // Dense accesses.
  {
    V v;
    VDBAccessor allocateWriteAccessor = v.allocateWriteAccessor();
    VDBAccessor writeAccessor = v.writeAccessor();
    VDBAccessor readAccessor = v.readAccessor();

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      allocateWriteAccessor.value(tec) = layout(tec);
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      AssertEq(allocateWriteAccessor.value(tec), layout(tec));
    }).Launch();

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable { writeAccessor.value(tec) *= -1; }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      AssertEq(writeAccessor.value(tec), -layout(tec));
    }).Launch();

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      AssertEq(readAccessor.value(tec), -layout(tec));
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable { writeAccessor.value(tec) += 233; }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      AssertEq(readAccessor.value(tec), -layout(tec) + 233);
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec, AllocateWriteAccessor accessor) mutable {
      accessor.value(tec) += 233;
    }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec, ReadAccessor accessor) mutable {
      AssertEq(accessor.value(tec), -layout(tec) + 466);
    }).Launch();
  }

  // Checkerboard accesses.
  {
    V v;
    VDBAccessor allocateWriteAccessor = v.allocateWriteAccessor();
    VDBAccessor writeAccessor = v.writeAccessor();
    VDBAccessor readAccessor = v.readAccessor();

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      if ((get<0>(tec) + get<1>(tec)) % 2 == 0)
        allocateWriteAccessor.value(tec) = layout(tec);
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      if ((get<0>(tec) + get<1>(tec)) % 2 == 0)
        AssertEq(allocateWriteAccessor.value(tec), layout(tec));
    }).Launch();

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      if ((get<0>(tec) + get<1>(tec)) % 2 == 0)
        writeAccessor.value(tec) *= -1;
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      if ((get<0>(tec) + get<1>(tec)) % 2 == 0)
        AssertEq(writeAccessor.value(tec), -layout(tec));
    }).Launch();

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      if ((get<0>(tec) + get<1>(tec)) % 2 == 0)
        AssertEq(readAccessor.value(tec), -layout(tec));
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable { writeAccessor.value(tec) += 233; }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      AssertEq(readAccessor.value(tec), -layout(tec) + 233);
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec, AllocateWriteAccessor &accessor) mutable {
      accessor.value(tec) += 233;
    }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec, const ReadAccessor &accessor) mutable {
      AssertEq(accessor.value(tec), -layout(tec) + 466);
    }).Launch();
  }

  // Sparse accesses, 1D.
  { // x.
    V v;
    VDBAccessor allocateWriteAccessor = v.allocateWriteAccessor();
    VDBAccessor writeAccessor = v.writeAccessor();
    VDBAccessor readAccessor = v.readAccessor();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable { allocateWriteAccessor.value({i - nHalf, 0}) = i - nHalf; }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(allocateWriteAccessor.value({i - nHalf, 0}), i - nHalf);
    }).Launch();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable { writeAccessor.value({i - nHalf, 0}) *= -2; }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(writeAccessor.value({i - nHalf, 0}), (i - nHalf) * (-2));
    }).Launch();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(readAccessor.value({i - nHalf, 0}), (i - nHalf) * (-2));
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable { writeAccessor.value(tec) *= (-3); }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      AssertEq(readAccessor.value(tec), get<0>(tec) * 6);
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec, WriteAccessor accessor) mutable {
      accessor.value(tec) *= (-3);
    }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec, ReadAccessor accessor) mutable {
      AssertEq(accessor.value(tec), get<0>(tec) * (-18));
    }).Launch();
  }
  { // y.
    V v;
    VDBAccessor allocateWriteAccessor = v.allocateWriteAccessor();
    VDBAccessor writeAccessor = v.writeAccessor();
    VDBAccessor readAccessor = v.readAccessor();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable { allocateWriteAccessor.value({0, i - nHalf}) = nHalf - i; }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(allocateWriteAccessor.value({0, i - nHalf}), nHalf - i);
    }).Launch();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable { writeAccessor.value({0, i - nHalf}) *= -2; }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(writeAccessor.value({0, i - nHalf}), (nHalf - i) * (-2));
    }).Launch();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(readAccessor.value({0, i - nHalf}), (nHalf - i) * (-2));
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable { writeAccessor.value(tec) *= (-3); }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      AssertEq(readAccessor.value(tec), get<1>(tec) * (-6));
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec, WriteAccessor &accessor) mutable {
      accessor.value(tec) *= (-3);
    }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec, ReadAccessor &accessor) mutable {
      AssertEq(accessor.value(tec), get<1>(tec) * 18);
    }).Launch();
  }

  // Sparse accesses, 2D.
  {
    V v;
    VDBAccessor allocateWriteAccessor = v.allocateWriteAccessor();
    VDBAccessor writeAccessor = v.writeAccessor();
    VDBAccessor readAccessor = v.readAccessor();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      allocateWriteAccessor.value({i - nHalf, i - nHalf}) = i - nHalf;
      allocateWriteAccessor.value({i - nHalf, nHalf - i}) = i - nHalf;
    }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(allocateWriteAccessor.value({i - nHalf, i - nHalf}), i - nHalf);
      AssertEq(allocateWriteAccessor.value({i - nHalf, nHalf - i}), i - nHalf);
    }).Launch();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      writeAccessor.value({i - nHalf, i - nHalf}) *= -3;
      if (i - nHalf != nHalf - i)
        writeAccessor.value({i - nHalf, nHalf - i}) *= -3;
    }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(writeAccessor.value({i - nHalf, i - nHalf}), (i - nHalf) * (-3));
      if (i - nHalf != nHalf - i)
        AssertEq(writeAccessor.value({i - nHalf, nHalf - i}), (i - nHalf) * (-3));
    }).Launch();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(readAccessor.value({i - nHalf, i - nHalf}), (i - nHalf) * (-3));
      if (i - nHalf != nHalf - i)
        AssertEq(readAccessor.value({i - nHalf, nHalf - i}), (i - nHalf) * (-3));
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable { writeAccessor.value(tec) *= (-2); }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(readAccessor.value({i - nHalf, i - nHalf}), (i - nHalf) * 6);
      if (i - nHalf != nHalf - i)
        AssertEq(readAccessor.value({i - nHalf, nHalf - i}), (i - nHalf) * 6);
    }).Launch();
  }

  cuda::device::current::get().synchronize();
}

template <typename T>
void Test3DVDBKernels() {
  using V = DeviceVDB<T, 3>;
  using AllocateWriteAccessor = VDBAllocateWriteAccessor<V>;
  using WriteAccessor = VDBWriteAccessor<V>;
  using ReadAccessor = VDBReadAccessor<V>;

  const Layout layout = make_layout_major(50, 100, 150);
  const int n = 1000;
  const int nHalf = n / 2;

  // CTAD.
  {
    V v;
    VDBAccessor allocateWriteAccessor = v.allocateWriteAccessor();
    VDBAccessor writeAccessor = v.writeAccessor();
    VDBAccessor readAccessor = v.readAccessor();

    static_assert(std::is_same_v<decltype(allocateWriteAccessor), AllocateWriteAccessor>);
    static_assert(std::is_same_v<decltype(writeAccessor), WriteAccessor>);
    static_assert(std::is_same_v<decltype(readAccessor), ReadAccessor>);
  }

  // Dense accesses.
  {
    V v;
    VDBAccessor allocateWriteAccessor = v.allocateWriteAccessor();
    VDBAccessor writeAccessor = v.writeAccessor();
    VDBAccessor readAccessor = v.readAccessor();

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      allocateWriteAccessor.value(tec) = layout(tec);
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      AssertEq(allocateWriteAccessor.value(tec), layout(tec));
    }).Launch();

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      writeAccessor.value(tec) *= -1;
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      AssertEq(writeAccessor.value(tec), -layout(tec));
    }).Launch();

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      AssertEq(readAccessor.value(tec), -layout(tec));
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable { writeAccessor.value(tec) += 233; }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      AssertEq(readAccessor.value(tec), -layout(tec) + 233);
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec, AllocateWriteAccessor accessor) mutable {
      accessor.value(tec) += 233;
    }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec, ReadAccessor accessor) mutable {
      AssertEq(accessor.value(tec), -layout(tec) + 466);
    }).Launch();
  }

  // Checkerboard accesses.
  {
    V v;
    VDBAccessor allocateWriteAccessor = v.allocateWriteAccessor();
    VDBAccessor writeAccessor = v.writeAccessor();
    VDBAccessor readAccessor = v.readAccessor();

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      if ((get<0>(tec) + get<1>(tec) + get<2>(tec)) % 2 == 0)
        allocateWriteAccessor.value(tec) = layout(tec);
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      if ((get<0>(tec) + get<1>(tec) + get<2>(tec)) % 2 == 0)
        AssertEq(allocateWriteAccessor.value(tec), layout(tec));
    }).Launch();

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      if ((get<0>(tec) + get<1>(tec) + get<2>(tec)) % 2 == 0)
        writeAccessor.value(tec) *= -1;
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      if ((get<0>(tec) + get<1>(tec) + get<2>(tec)) % 2 == 0)
        AssertEq(writeAccessor.value(tec), -layout(tec));
    }).Launch();

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      if ((get<0>(tec) + get<1>(tec) + get<2>(tec)) % 2 == 0)
        AssertEq(readAccessor.value(tec), -layout(tec));
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable { writeAccessor.value(tec) += 233; }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      AssertEq(readAccessor.value(tec), -layout(tec) + 233);
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec, AllocateWriteAccessor &accessor) mutable {
      accessor.value(tec) += 233;
    }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec, const ReadAccessor &accessor) mutable {
      AssertEq(accessor.value(tec), -layout(tec) + 466);
    }).Launch();
  }

  // Sparse accesses, 1D.
  { // x.
    V v;
    VDBAccessor allocateWriteAccessor = v.allocateWriteAccessor();
    VDBAccessor writeAccessor = v.writeAccessor();
    VDBAccessor readAccessor = v.readAccessor();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      allocateWriteAccessor.value({i - nHalf, 0, 0}) = i - nHalf;
    }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(allocateWriteAccessor.value({i - nHalf, 0, 0}), i - nHalf);
    }).Launch();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable { writeAccessor.value({i - nHalf, 0, 0}) *= -2; }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(writeAccessor.value({i - nHalf, 0, 0}), (i - nHalf) * (-2));
    }).Launch();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(readAccessor.value({i - nHalf, 0, 0}), (i - nHalf) * (-2));
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable { writeAccessor.value(tec) *= (-3); }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      AssertEq(readAccessor.value(tec), get<0>(tec) * 6);
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec, WriteAccessor accessor) mutable {
      accessor.value(tec) *= (-3);
    }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec, ReadAccessor accessor) mutable {
      AssertEq(accessor.value(tec), get<0>(tec) * (-18));
    }).Launch();
  }
  { // y.
    V v;
    VDBAccessor allocateWriteAccessor = v.allocateWriteAccessor();
    VDBAccessor writeAccessor = v.writeAccessor();
    VDBAccessor readAccessor = v.readAccessor();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      allocateWriteAccessor.value({0, i - nHalf, 0}) = nHalf - i;
    }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(allocateWriteAccessor.value({0, i - nHalf, 0}), nHalf - i);
    }).Launch();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable { writeAccessor.value({0, i - nHalf, 0}) *= -2; }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(writeAccessor.value({0, i - nHalf, 0}), (nHalf - i) * (-2));
    }).Launch();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(readAccessor.value({0, i - nHalf, 0}), (nHalf - i) * (-2));
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable { writeAccessor.value(tec) *= (-3); }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      AssertEq(readAccessor.value(tec), get<1>(tec) * (-6));
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec, WriteAccessor &accessor) mutable {
      accessor.value(tec) *= (-3);
    }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec, ReadAccessor &accessor) mutable {
      AssertEq(accessor.value(tec), get<1>(tec) * 18);
    }).Launch();
  }
  { // z.
    V v;
    VDBAccessor allocateWriteAccessor = v.allocateWriteAccessor();
    VDBAccessor writeAccessor = v.writeAccessor();
    VDBAccessor readAccessor = v.readAccessor();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      allocateWriteAccessor.value({0, 0, i - nHalf}) = nHalf - i;
    }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(allocateWriteAccessor.value({0, 0, i - nHalf}), nHalf - i);
    }).Launch();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable { writeAccessor.value({0, 0, i - nHalf}) *= -2; }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(writeAccessor.value({0, 0, i - nHalf}), (nHalf - i) * (-2));
    }).Launch();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(readAccessor.value({0, 0, i - nHalf}), (nHalf - i) * (-2));
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable { writeAccessor.value(tec) *= (-3); }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      AssertEq(readAccessor.value(tec), get<2>(tec) * (-6));
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec, AllocateWriteAccessor &accessor) mutable {
      accessor.value(tec) *= (-3);
    }).Launch();
    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec, const AllocateWriteAccessor &accessor) mutable {
      AssertEq(accessor.value(tec), get<2>(tec) * 18);
    }).Launch();
  }

  // Sparse accesses, 3D.
  {
    V v;
    VDBAccessor allocateWriteAccessor = v.allocateWriteAccessor();
    VDBAccessor writeAccessor = v.writeAccessor();
    VDBAccessor readAccessor = v.readAccessor();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      allocateWriteAccessor.value({i - nHalf, i - nHalf, 0}) = i - nHalf;
      allocateWriteAccessor.value({i - nHalf, nHalf - i, 0}) = i - nHalf;

      if (i - nHalf != nHalf - i) {
        allocateWriteAccessor.value({i - nHalf, 0, i - nHalf}) = (i - nHalf) * 2;
        allocateWriteAccessor.value({i - nHalf, 0, nHalf - i}) = (i - nHalf) * 2;
      }

      if (i - nHalf != nHalf - i) {
        allocateWriteAccessor.value({0, i - nHalf, i - nHalf}) = (i - nHalf) * (-3);
        allocateWriteAccessor.value({0, i - nHalf, nHalf - i}) = (i - nHalf) * (-3);
      }
    }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(allocateWriteAccessor.value({i - nHalf, i - nHalf, 0}), i - nHalf);
      AssertEq(allocateWriteAccessor.value({i - nHalf, nHalf - i, 0}), i - nHalf);

      if (i - nHalf != nHalf - i) {
        AssertEq(allocateWriteAccessor.value({i - nHalf, 0, i - nHalf}), (i - nHalf) * 2);
        AssertEq(allocateWriteAccessor.value({i - nHalf, 0, nHalf - i}), (i - nHalf) * 2);
      }

      if (i - nHalf != nHalf - i) {
        AssertEq(allocateWriteAccessor.value({0, i - nHalf, i - nHalf}), (i - nHalf) * (-3));
        AssertEq(allocateWriteAccessor.value({0, i - nHalf, nHalf - i}), (i - nHalf) * (-3));
      }
    }).Launch();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      allocateWriteAccessor.value({i - nHalf, i - nHalf, 0}) = i - nHalf;
      allocateWriteAccessor.value({i - nHalf, nHalf - i, 0}) = i - nHalf;

      allocateWriteAccessor.value({i - nHalf, 0, i - nHalf}) = i - nHalf;
      allocateWriteAccessor.value({i - nHalf, 0, nHalf - i}) = i - nHalf;

      allocateWriteAccessor.value({0, i - nHalf, i - nHalf}) = i - nHalf;
      allocateWriteAccessor.value({0, i - nHalf, nHalf - i}) = i - nHalf;
    }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      writeAccessor.value({i - nHalf, i - nHalf, 0}) *= -3;
      if (i - nHalf != nHalf - i)
        writeAccessor.value({i - nHalf, nHalf - i, 0}) *= -3;

      writeAccessor.value({i - nHalf, 0, i - nHalf}) *= -3;
      if (i - nHalf != nHalf - i)
        writeAccessor.value({i - nHalf, 0, nHalf - i}) *= -3;

      writeAccessor.value({0, i - nHalf, i - nHalf}) *= -3;
      if (i - nHalf != nHalf - i)
        writeAccessor.value({0, i - nHalf, nHalf - i}) *= -3;
    }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(writeAccessor.value({i - nHalf, i - nHalf, 0}), (i - nHalf) * (-3));
      if (i - nHalf != nHalf - i)
        AssertEq(writeAccessor.value({i - nHalf, nHalf - i, 0}), (i - nHalf) * (-3));

      AssertEq(writeAccessor.value({i - nHalf, 0, i - nHalf}), (i - nHalf) * (-3));
      if (i - nHalf != nHalf - i)
        AssertEq(writeAccessor.value({i - nHalf, 0, nHalf - i}), (i - nHalf) * (-3));

      AssertEq(writeAccessor.value({0, i - nHalf, i - nHalf}), (i - nHalf) * (-3));
      if (i - nHalf != nHalf - i)
        AssertEq(writeAccessor.value({0, i - nHalf, nHalf - i}), (i - nHalf) * (-3));
    }).Launch();

    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(writeAccessor.value({i - nHalf, i - nHalf, 0}), (i - nHalf) * (-3));
      if (i - nHalf != nHalf - i)
        AssertEq(writeAccessor.value({i - nHalf, nHalf - i, 0}), (i - nHalf) * (-3));

      AssertEq(writeAccessor.value({i - nHalf, 0, i - nHalf}), (i - nHalf) * (-3));
      if (i - nHalf != nHalf - i)
        AssertEq(writeAccessor.value({i - nHalf, 0, nHalf - i}), (i - nHalf) * (-3));

      AssertEq(writeAccessor.value({0, i - nHalf, i - nHalf}), (i - nHalf) * (-3));
      if (i - nHalf != nHalf - i)
        AssertEq(writeAccessor.value({0, i - nHalf, nHalf - i}), (i - nHalf) * (-3));
    }).Launch();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable { writeAccessor.value(tec) *= (-2); }).Launch();
    Launcher(n, [=] ARIA_DEVICE(int i) mutable {
      AssertEq(writeAccessor.value({i - nHalf, i - nHalf, 0}), (i - nHalf) * 6);
      if (i - nHalf != nHalf - i)
        AssertEq(writeAccessor.value({i - nHalf, nHalf - i, 0}), (i - nHalf) * 6);

      AssertEq(writeAccessor.value({i - nHalf, 0, i - nHalf}), (i - nHalf) * 6);
      if (i - nHalf != nHalf - i)
        AssertEq(writeAccessor.value({i - nHalf, 0, nHalf - i}), (i - nHalf) * 6);

      AssertEq(writeAccessor.value({0, i - nHalf, i - nHalf}), (i - nHalf) * 6);
      if (i - nHalf != nHalf - i)
        AssertEq(writeAccessor.value({0, i - nHalf, nHalf - i}), (i - nHalf) * 6);
    }).Launch();
  }

  cuda::device::current::get().synchronize();
}

template <typename T>
void Test1DVDBSetOffAndShrinkKernels() {
  using V = DeviceVDB<T, 1>;

  const Layout layout = make_layout_major(20000);
  const int n = size(layout);

  thrust::device_vector<int> counterD(1);
  thrust::device_ptr counter = counterD.data();

  // Dense off accesses.
  {
    V v;

    VDBAccessor accessor = v.allocateWriteAccessor();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int> &tec) mutable { ARIA_ASSERT(!accessor.IsValueOn(tec)); }).Launch();
    v.ShrinkToFit();

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int> &tec) mutable { accessor.value(tec) = 0; }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int> &tec) mutable { ARIA_ASSERT(accessor.IsValueOn(tec)); }).Launch();
    v.ShrinkToFit();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int> &tec) mutable {
      accessor.value(tec) = Off{};
      atomicAdd(counter.get(), 1);
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int> &tec) mutable { ARIA_ASSERT(!accessor.IsValueOn(tec)); }).Launch();
    cuda::device::current::get().synchronize();
    EXPECT_EQ(*counter, n);
    *counter = 0;

    for (int round = 0; round < 3; ++round) {
      Launcher(v, [=] ARIA_DEVICE(const Tec<int> &tec) mutable { ARIA_ASSERT(false); }).Launch();
      v.ShrinkToFit();
    }

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int> &tec) mutable { accessor.value(tec) = Off{}; }).Launch();
    for (int round = 0; round < 3; ++round) {
      Launcher(v, [=] ARIA_DEVICE(const Tec<int> &tec) mutable { ARIA_ASSERT(false); }).Launch();
      v.ShrinkToFit();
    }
  }

  // Checkerboard accesses.
  {
    V v;

    VDBAccessor accessor = v.allocateWriteAccessor();
    v.ShrinkToFit();

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int> &tec) mutable { accessor.value(tec) = 0; }).Launch();
    v.ShrinkToFit();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int> &tec) mutable {
      if (get<0>(tec) % 2 == 0) {
        accessor.value(tec) = Off{};
        atomicAdd(counter.get(), 1);
      }
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int> &tec) mutable {
      if (get<0>(tec) % 2 == 0)
        ARIA_ASSERT(!accessor.IsValueOn(tec));
      else
        ARIA_ASSERT(accessor.IsValueOn(tec));
    }).Launch();
    cuda::device::current::get().synchronize();
    EXPECT_EQ(*counter, n / 2);
    *counter = 0;

    for (int round = 0; round < 3; ++round) {
      Launcher(v, [=] ARIA_DEVICE(const Tec<int> &tec) mutable {
        if (get<0>(tec) % 2 == 0)
          ARIA_ASSERT(false);
        else
          atomicAdd(counter.get(), 1);
      }).Launch();
      cuda::device::current::get().synchronize();
      EXPECT_EQ(*counter, n / 2);
      *counter = 0;
      v.ShrinkToFit();
    }

    Launcher(v, [=] ARIA_DEVICE(const Tec<int> &tec) mutable {
      accessor.value(tec) = Off{};
      atomicAdd(counter.get(), 1);
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int> &tec) mutable { ARIA_ASSERT(!accessor.IsValueOn(tec)); }).Launch();
    cuda::device::current::get().synchronize();
    EXPECT_EQ(*counter, n / 2);
    *counter = 0;

    for (int round = 0; round < 3; ++round) {
      Launcher(v, [=] ARIA_DEVICE(const Tec<int> &tec) mutable { ARIA_ASSERT(false); }).Launch();
      v.ShrinkToFit();
    }
  }
}

template <typename T>
void Test2DVDBSetOffAndShrinkKernels() {
  using V = DeviceVDB<T, 2>;

  const Layout layout = make_layout_major(200, 300);
  const int n = size(layout);

  thrust::device_vector<int> counterD(1);
  thrust::device_ptr counter = counterD.data();

  // Dense off accesses.
  {
    V v;

    VDBAccessor accessor = v.allocateWriteAccessor();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      ARIA_ASSERT(!accessor.IsValueOn(tec));
    }).Launch();
    v.ShrinkToFit();

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable { accessor.value(tec) = 0; }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      ARIA_ASSERT(accessor.IsValueOn(tec));
    }).Launch();
    v.ShrinkToFit();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      accessor.value(tec) = Off{};
      atomicAdd(counter.get(), 1);
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      ARIA_ASSERT(!accessor.IsValueOn(tec));
    }).Launch();
    cuda::device::current::get().synchronize();
    EXPECT_EQ(*counter, n);
    *counter = 0;

    for (int round = 0; round < 3; ++round) {
      Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable { ARIA_ASSERT(false); }).Launch();
      v.ShrinkToFit();
    }

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable { accessor.value(tec) = Off{}; }).Launch();
    for (int round = 0; round < 3; ++round) {
      Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable { ARIA_ASSERT(false); }).Launch();
      v.ShrinkToFit();
    }
  }

  // Checkerboard accesses.
  {
    V v;

    VDBAccessor accessor = v.allocateWriteAccessor();
    v.ShrinkToFit();

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable { accessor.value(tec) = 0; }).Launch();
    v.ShrinkToFit();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      if ((get<0>(tec) + get<1>(tec)) % 2 == 0) {
        accessor.value(tec) = Off{};
        atomicAdd(counter.get(), 1);
      }
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      if ((get<0>(tec) + get<1>(tec)) % 2 == 0)
        ARIA_ASSERT(!accessor.IsValueOn(tec));
      else
        ARIA_ASSERT(accessor.IsValueOn(tec));
    }).Launch();
    cuda::device::current::get().synchronize();
    EXPECT_EQ(*counter, n / 2);
    *counter = 0;

    for (int round = 0; round < 3; ++round) {
      Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
        if ((get<0>(tec) + get<1>(tec)) % 2 == 0)
          ARIA_ASSERT(false);
        else
          atomicAdd(counter.get(), 1);
      }).Launch();
      cuda::device::current::get().synchronize();
      EXPECT_EQ(*counter, n / 2);
      *counter = 0;
      v.ShrinkToFit();
    }

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      accessor.value(tec) = Off{};
      atomicAdd(counter.get(), 1);
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable {
      ARIA_ASSERT(!accessor.IsValueOn(tec));
    }).Launch();
    cuda::device::current::get().synchronize();
    EXPECT_EQ(*counter, n / 2);
    *counter = 0;

    for (int round = 0; round < 3; ++round) {
      Launcher(v, [=] ARIA_DEVICE(const Tec<int, int> &tec) mutable { ARIA_ASSERT(false); }).Launch();
      v.ShrinkToFit();
    }
  }
}

template <typename T>
void Test3DVDBSetOffAndShrinkKernels() {
  using V = DeviceVDB<T, 3>;

  const Layout layout = make_layout_major(50, 100, 150);
  const int n = size(layout);

  thrust::device_vector<int> counterD(1);
  thrust::device_ptr counter = counterD.data();

  // Dense off accesses.
  {
    V v;

    VDBAccessor accessor = v.allocateWriteAccessor();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      ARIA_ASSERT(!accessor.IsValueOn(tec));
    }).Launch();
    v.ShrinkToFit();

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable { accessor.value(tec) = 0; }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      ARIA_ASSERT(accessor.IsValueOn(tec));
    }).Launch();
    v.ShrinkToFit();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      accessor.value(tec) = Off{};
      atomicAdd(counter.get(), 1);
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      ARIA_ASSERT(!accessor.IsValueOn(tec));
    }).Launch();
    cuda::device::current::get().synchronize();
    EXPECT_EQ(*counter, n);
    *counter = 0;

    for (int round = 0; round < 3; ++round) {
      Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable { ARIA_ASSERT(false); }).Launch();
      v.ShrinkToFit();
    }

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable { accessor.value(tec) = Off{}; }).Launch();
    for (int round = 0; round < 3; ++round) {
      Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable { ARIA_ASSERT(false); }).Launch();
      v.ShrinkToFit();
    }
  }

  // Checkerboard accesses.
  {
    V v;

    VDBAccessor accessor = v.allocateWriteAccessor();
    v.ShrinkToFit();

    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable { accessor.value(tec) = 0; }).Launch();
    v.ShrinkToFit();

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      if ((get<0>(tec) + get<1>(tec) + get<2>(tec)) % 2 == 0) {
        accessor.value(tec) = Off{};
        atomicAdd(counter.get(), 1);
      }
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      if ((get<0>(tec) + get<1>(tec) + get<2>(tec)) % 2 == 0)
        ARIA_ASSERT(!accessor.IsValueOn(tec));
      else
        ARIA_ASSERT(accessor.IsValueOn(tec));
    }).Launch();
    cuda::device::current::get().synchronize();
    EXPECT_EQ(*counter, n / 2);
    *counter = 0;

    for (int round = 0; round < 3; ++round) {
      Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
        if ((get<0>(tec) + get<1>(tec) + get<2>(tec)) % 2 == 0)
          ARIA_ASSERT(false);
        else
          atomicAdd(counter.get(), 1);
      }).Launch();
      cuda::device::current::get().synchronize();
      EXPECT_EQ(*counter, n / 2);
      *counter = 0;
      v.ShrinkToFit();
    }

    Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      accessor.value(tec) = Off{};
      atomicAdd(counter.get(), 1);
    }).Launch();
    Launcher(layout, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable {
      ARIA_ASSERT(!accessor.IsValueOn(tec));
    }).Launch();
    cuda::device::current::get().synchronize();
    EXPECT_EQ(*counter, n / 2);
    *counter = 0;

    for (int round = 0; round < 3; ++round) {
      Launcher(v, [=] ARIA_DEVICE(const Tec<int, int, int> &tec) mutable { ARIA_ASSERT(false); }).Launch();
      v.ShrinkToFit();
    }
  }
}

} // namespace

TEST(VDB, Base) {
  size_t size = 1LLU * 1024LLU * 1024LLU * 1024LLU; // 1GB
  cuda::device::current::get().set_limit(hipLimitMallocHeapSize, size);

  using T = float;
  using TMosaic = Mosaic<T, PatternFloats>;

  auto testVDBBase = []<typename T, auto dim>() {
    using V = DeviceVDB<T, dim>;
    using AllocateWriteAccessor = VDBAllocateWriteAccessor<V>;
    using WriteAccessor = VDBWriteAccessor<V>;
    using ReadAccessor = VDBReadAccessor<V>;

    // Constructors.
    V v0{};

    // Move.
    V v = std::move(v0);

    // Allocate-write accessor.
    {
      // Constructors.
      AllocateWriteAccessor accessor0;
      AllocateWriteAccessor accessor1 = v.allocateWriteAccessor();

      // Copy.
      AllocateWriteAccessor accessor2 = accessor0;
      AllocateWriteAccessor accessor3 = accessor1;

      // Move.
      AllocateWriteAccessor accessor4 = std::move(accessor0);
      AllocateWriteAccessor accessor5 = std::move(accessor1);

      // Destructor.
    }

    // Write accessor.
    {
      // Constructors.
      WriteAccessor accessor0;
      WriteAccessor accessor1 = v.writeAccessor();

      // Copy.
      WriteAccessor accessor2 = accessor0;
      WriteAccessor accessor3 = accessor1;

      // Move.
      WriteAccessor accessor4 = std::move(accessor0);
      WriteAccessor accessor5 = std::move(accessor1);

      // Destructor.
    }

    // Read accessor.
    {
      // Constructors.
      ReadAccessor accessor0;
      ReadAccessor accessor1 = v.readAccessor();

      // Copy.
      ReadAccessor accessor2 = accessor0;
      ReadAccessor accessor3 = accessor1;

      // Move.
      ReadAccessor accessor4 = std::move(accessor0);
      ReadAccessor accessor5 = std::move(accessor1);

      // Destructor.
    }

    // Destructor.
  };

  testVDBBase.operator()<T, 1>();
  testVDBBase.operator()<TMosaic, 1>();
  testVDBBase.operator()<T, 2>();
  testVDBBase.operator()<TMosaic, 2>();
  testVDBBase.operator()<T, 3>();
  testVDBBase.operator()<TMosaic, 3>();
  testVDBBase.operator()<T, 4>();
  testVDBBase.operator()<TMosaic, 4>();
}

TEST(VDB, Handle) {
  using T = float;
  using TMosaic = Mosaic<T, PatternFloats>;

  auto testVDBHandleBase = []<typename T, auto dim>() {
    using Handle = vdb::detail::VDBHandle<T, dim, SpaceDevice>;

    // Constructors and create.
    Handle handle0;
    Handle handle1 = Handle::Create();

    // Copy.
    Handle handle2 = handle0;
    Handle handle3 = handle1;

    // Move.
    Handle handle4 = std::move(handle0);
    Handle handle5 = std::move(handle1);

    // Destructor and destroy.
    handle1.Destroy();
  };

  testVDBHandleBase.operator()<T, 1>();
  testVDBHandleBase.operator()<TMosaic, 1>();
  testVDBHandleBase.operator()<T, 2>();
  testVDBHandleBase.operator()<TMosaic, 2>();
  testVDBHandleBase.operator()<T, 3>();
  testVDBHandleBase.operator()<TMosaic, 3>();
  testVDBHandleBase.operator()<T, 4>();
  testVDBHandleBase.operator()<TMosaic, 4>();

  Test1DVDBHandleKernels<T>();
  Test1DVDBHandleKernels<TMosaic>();
  Test2DVDBHandleKernels<T>();
  Test2DVDBHandleKernels<TMosaic>();
  Test3DVDBHandleKernels<T>();
  Test3DVDBHandleKernels<TMosaic>();
}

TEST(VDB, VDB) {
  using T = int;
  using TMosaic = Mosaic<T, PatternInts>;

  Test1DVDBKernels<T>();
  Test1DVDBKernels<TMosaic>();
  Test2DVDBKernels<T>();
  Test2DVDBKernels<TMosaic>();
  Test3DVDBKernels<T>();
  Test3DVDBKernels<TMosaic>();
}

TEST(VDB, SetOffAndShrink) {
  using T = int;
  using TMosaic = Mosaic<T, PatternInts>;

  Test1DVDBSetOffAndShrinkKernels<T>();
  Test1DVDBSetOffAndShrinkKernels<TMosaic>();
  Test2DVDBSetOffAndShrinkKernels<T>();
  Test2DVDBSetOffAndShrinkKernels<TMosaic>();
  Test3DVDBSetOffAndShrinkKernels<T>();
  Test3DVDBSetOffAndShrinkKernels<TMosaic>();
}

} // namespace ARIA
