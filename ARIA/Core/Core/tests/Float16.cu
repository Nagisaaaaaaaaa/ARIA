#include "hip/hip_runtime.h"
#include "ARIA/Float16.h"

#include <cuda/api.hpp>
#include <gtest/gtest.h>

namespace ARIA {

namespace {

ARIA_KERNEL void TestBaseCUDAKernel() {
  ARIA_ASSERT(float16{} == HIPRT_ZERO_FP16);

  float16 a{0.1F};
  float16 b{0.2F};
  float16 c = a + b;
  ARIA_ASSERT(std::abs(static_cast<float>(c) - 0.3F) < 0.00025F);

  ARIA_ASSERT(std::numeric_limits<float16>::min() > float16{});
  ARIA_ASSERT(std::numeric_limits<float16>::max() > float16{});
  ARIA_ASSERT(static_cast<float>(std::numeric_limits<float16>::infinity()) == std::numeric_limits<float>::infinity());
  ARIA_ASSERT(static_cast<float>(-std::numeric_limits<float16>::infinity()) == -std::numeric_limits<float>::infinity());
  ARIA_ASSERT(std::numeric_limits<float16>::quiet_NaN() != std::numeric_limits<float16>::quiet_NaN());

  ARIA_ASSERT(cuda::std::numeric_limits<float16>::min() > float16{});
  ARIA_ASSERT(cuda::std::numeric_limits<float16>::max() > float16{});
  ARIA_ASSERT(static_cast<float>(cuda::std::numeric_limits<float16>::infinity()) ==
              cuda::std::numeric_limits<float>::infinity());
  ARIA_ASSERT(static_cast<float>(-cuda::std::numeric_limits<float16>::infinity()) ==
              -cuda::std::numeric_limits<float>::infinity());
  ARIA_ASSERT(cuda::std::numeric_limits<float16>::quiet_NaN() != cuda::std::numeric_limits<float16>::quiet_NaN());
}

ARIA_KERNEL void TestMathCUDAKernel() {
  float16 a{0.1F};
  float16 b{-0.1F};

  ARIA_ASSERT(abs(a) == a);
  ARIA_ASSERT(abs(b) == a);
  ARIA_ASSERT(std::abs(a) == a);
  ARIA_ASSERT(std::abs(b) == a);
  ARIA_ASSERT(cuda::std::abs(a) == a);
  ARIA_ASSERT(cuda::std::abs(b) == a);

  ARIA_ASSERT(max(a, b) == a);
  ARIA_ASSERT(min(a, b) == b);
  ARIA_ASSERT(std::max(a, b) == a);
  ARIA_ASSERT(std::min(a, b) == b);
  ARIA_ASSERT(cuda::std::max(a, b) == a);
  ARIA_ASSERT(cuda::std::min(a, b) == b);
}

void TestBaseCUDA() {
  try {
    TestBaseCUDAKernel<<<1, 1>>>();
    cuda::device::current::get().synchronize();
  } catch (...) { EXPECT_FALSE(true); }
}

void TestMathCUDA() {
  try {
    TestMathCUDAKernel<<<1, 1>>>();
    cuda::device::current::get().synchronize();
  } catch (...) { EXPECT_FALSE(true); }
}

} // namespace

TEST(Float16, Base) {
  EXPECT_EQ(float16{}, HIPRT_ZERO_FP16);

  float16 a{0.1F};
  float16 b{0.2F};
  float16 c = a + b;
  EXPECT_TRUE(std::abs(static_cast<float>(c) - 0.3F) < 0.00025F);

  EXPECT_TRUE(std::numeric_limits<float16>::min() > float16{});
  EXPECT_TRUE(std::numeric_limits<float16>::max() > float16{});
  EXPECT_TRUE(static_cast<float>(std::numeric_limits<float16>::infinity()) == std::numeric_limits<float>::infinity());
  EXPECT_TRUE(static_cast<float>(-std::numeric_limits<float16>::infinity()) == -std::numeric_limits<float>::infinity());
  EXPECT_NE(std::numeric_limits<float16>::quiet_NaN(), std::numeric_limits<float16>::quiet_NaN());

  EXPECT_TRUE(cuda::std::numeric_limits<float16>::min() > float16{});
  EXPECT_TRUE(cuda::std::numeric_limits<float16>::max() > float16{});
  EXPECT_TRUE(static_cast<float>(cuda::std::numeric_limits<float16>::infinity()) ==
              cuda::std::numeric_limits<float>::infinity());
  EXPECT_TRUE(static_cast<float>(-cuda::std::numeric_limits<float16>::infinity()) ==
              -cuda::std::numeric_limits<float>::infinity());
  EXPECT_NE(cuda::std::numeric_limits<float16>::quiet_NaN(), cuda::std::numeric_limits<float16>::quiet_NaN());

  TestBaseCUDA();
}

TEST(Float16, Math) {
  float16 a{0.1F};
  float16 b{-0.1F};

  EXPECT_EQ(abs(a), a);
  EXPECT_EQ(abs(b), a);
  EXPECT_EQ(std::abs(a), a);
  EXPECT_EQ(std::abs(b), a);
  EXPECT_EQ(cuda::std::abs(a), a);
  EXPECT_EQ(cuda::std::abs(b), a);

  EXPECT_EQ(max(a, b), a);
  EXPECT_EQ(min(a, b), b);
  EXPECT_EQ(std::max(a, b), a);
  EXPECT_EQ(std::min(a, b), b);
  EXPECT_EQ(cuda::std::max(a, b), a);
  EXPECT_EQ(cuda::std::min(a, b), b);

  TestMathCUDA();
}

TEST(Float16, Python) {
  Python::ScopedInterpreter guard{};

  Python::Module main = guard.Import("__main__");
  Python::Dict local{main};

  float16 a{0.1F};
  float16 b{0.2F};
  float16 a_add_b = a + b;
  float16 a_sub_b = a - b;
  float16 a_mul_b = a * b;
  float16 a_div_b = a / b;

  local["a_copy"] = a;
  local["minusA_copy"] = -a;
  local["a_add_b"] = a_add_b;
  local["a_sub_b"] = &a_sub_b;
  local["a_mul_b"] = a_mul_b;
  local["a_div_b"] = &a_div_b;

  try {
    py::exec("a = float16(0.1)\n"
             "b = float16(0.2)\n"
             "c = float16(233)\n"
             "\n"
             "assert a < b\n"
             "assert b > a\n"
             "assert a <= b\n"
             "assert b >= a\n"
             "assert +a == a_copy\n"
             "assert -a == minusA_copy\n"
             "assert a + b == a_add_b\n"
             "assert a - b == a_sub_b\n"
             "assert a * b == a_mul_b\n"
             "assert a / b == a_div_b\n"
             "\n"
             "assert abs(-a) == a\n"
             "assert max(a, b) == b\n"
             "assert min(a, b) == a\n",
             py::globals(), local);
  } catch (std::exception &e) {
    fmt::print("{}\n", e.what());
    EXPECT_FALSE(true);
  }
}

} // namespace ARIA
